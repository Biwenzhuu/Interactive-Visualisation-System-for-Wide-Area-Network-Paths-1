#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2\opencv.hpp>
#include <iostream>
using namespace std;
using namespace cv;

//Sobel算子边缘检测核函数
__global__ void sobelInCuda(unsigned char *dataIn, unsigned char *dataOut, int imgHeight, int imgWidth)
{
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    int index = yIndex * imgWidth + xIndex;
    int Gx = 0;
    int Gy = 0;

    if (xIndex > 0 && xIndex < imgWidth - 1 && yIndex > 0 && yIndex < imgHeight - 1)
    {
        Gx = dataIn[(yIndex - 1) * imgWidth + xIndex + 1] + 2 * dataIn[yIndex * imgWidth + xIndex + 1] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[yIndex * imgWidth + xIndex - 1] + dataIn[(yIndex + 1) * imgWidth + xIndex - 1]);
        Gy = dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex - 1) * imgWidth + xIndex] + dataIn[(yIndex - 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex + 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex + 1) * imgWidth + xIndex] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]);
        dataOut[index] = (abs(Gx) + abs(Gy)) / 2;
    }
}

//Sobel算子边缘检测CPU函数
void sobel(Mat srcImg, Mat dstImg, int imgHeight, int imgWidth)
{
    int Gx = 0;
    int Gy = 0;
    for (int i = 1; i < imgHeight - 1; i++)
    {
        uchar *dataUp = srcImg.ptr<uchar>(i - 1);
        uchar *data = srcImg.ptr<uchar>(i);
        uchar *dataDown = srcImg.ptr<uchar>(i + 1);
        uchar *out = dstImg.ptr<uchar>(i);
        for (int j = 1; j < imgWidth - 1; j++)
        {
            Gx = (dataUp[j + 1] + 2 * data[j + 1] + dataDown[j + 1]) - (dataUp[j - 1] + 2 * data[j - 1] + dataDown[j - 1]);
            Gy = (dataUp[j - 1] + 2 * dataUp[j] + dataUp[j + 1]) - (dataDown[j - 1] + 2 * dataDown[j] + dataDown[j + 1]);
            out[j] = (abs(Gx) + abs(Gy)) / 2;
        }
    }
}

int main()
{
    Mat grayImg = imread("1.jpg", 0);

    int imgHeight = grayImg.rows;
    int imgWidth = grayImg.cols;

    Mat gaussImg;
    //高斯滤波
    GaussianBlur(grayImg, gaussImg, Size(3, 3), 0, 0, BORDER_DEFAULT);

    //Sobel算子CPU实现
    Mat dst(imgHeight, imgWidth, CV_8UC1, Scalar(0));
    sobel(gaussImg, dst, imgHeight, imgWidth);

    //CUDA实现后的传回的图像
    Mat dstImg(imgHeight, imgWidth, CV_8UC1, Scalar(0));

    //创建GPU内存
    unsigned char *d_in;
    unsigned char *d_out;

    hipMalloc((void**)&d_in, imgHeight * imgWidth * sizeof(unsigned char));
    hipMalloc((void**)&d_out, imgHeight * imgWidth * sizeof(unsigned char));

    //将高斯滤波后的图像从CPU传入GPU
    hipMemcpy(d_in, gaussImg.data, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (imgHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);

    //调用核函数
    sobelInCuda << <blocksPerGrid, threadsPerBlock >> >(d_in, d_out, imgHeight, imgWidth);

    //将图像传回GPU
    hipMemcpy(dstImg.data, d_out, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //释放GPU内存
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
